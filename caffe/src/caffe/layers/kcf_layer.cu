#include <algorithm>
#include <vector>

#include "caffe/layers/kcf_layer.hpp"
#include "caffe/util/math_functions.hpp"
// #include "hipfft/hipfft.h"

namespace caffe {

template <typename Dtype>
void KCFLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  top[0]->mutable_cpu_data()[0] = Dtype(1.0);
  return;
}

template <typename Dtype>
void KCFLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  return;
}

INSTANTIATE_LAYER_GPU_FUNCS(KCFLayer);

}  // namespace caffe
