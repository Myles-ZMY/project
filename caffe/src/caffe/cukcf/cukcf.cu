#include "hip/hip_runtime.h"
#include <math_functions.h>
#include <thrust/device_vector.h>
#include <thrust/functional.h>  // thrust::plus
#include <thrust/reduce.h>

#include "caffe/common.hpp"
#include "caffe/cukcf/cukcf.hpp"
#include "caffe/util/math_functions.hpp"
namespace caffe {

__global__ void conj_mul_kernel(const int n, const hipComplex* a,
		const hipComplex* b, hipComplex* dst) {
	CUDA_KERNEL_LOOP(index, n) {
		dst[index] = hipCmulf(hipConjf(a[index]), b[index]);
	}
}

void caffe_gpu_conj_mul(const int N, const hipComplex* a, const hipComplex* b,
		hipComplex* dst) {
	conj_mul_kernel<<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
			N, a, b, dst);
}

__global__ void add_scalar_C_kernel(const int n, const hipComplex* a,
		const hipComplex alpha, hipComplex* dst) {
	CUDA_KERNEL_LOOP(index, n) {
		dst[index] = hipCaddf(a[index], alpha);
	}
}

void caffe_gpu_add_scalar_C(const int N, const hipComplex* a, const hipComplex alpha, 
		hipComplex* dst) {
	add_scalar_C_kernel<<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
			N, a, alpha, dst);
}


template <>
void caffe_gpu_gemv<hipComplex>(const CBLAS_TRANSPOSE TransA, const int M,
		const int N, const hipComplex alpha, const hipComplex* A, const hipComplex* x,
		const hipComplex beta, hipComplex* y) {
	hipblasOperation_t cuTransA =
		(TransA == CblasNoTrans) ? HIPBLAS_OP_T : HIPBLAS_OP_N;
	CUBLAS_CHECK(hipblasCgemv(Caffe::cublas_handle(), cuTransA, N, M, &alpha,
		A, N, x, 1, &beta, y, 1));
}

}
