#include "hip/hip_runtime.h"
#include <math_functions.h>
#include <thrust/device_vector.h>
#include <thrust/functional.h>  // thrust::plus
#include <thrust/reduce.h>

#include "caffe/common.hpp"
#include "caffe/cukcf/cuTracker.hpp"
#include "caffe/util/math_functions.hpp"
namespace caffe {
__global__ void set_C_kernel(const int n, const hipComplex alpha, hipComplex* dst) {
	CUDA_KERNEL_LOOP(index, n) {
		dst[index] = alpha;
	}
}

void caffe_gpu_set_C(const int N, const hipComplex alpha, hipComplex* dst) {
	real_C_kernel<<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
			N, alpha, dst);
}

__global__ void real_C_kernel(const int n, const hipComplex* a, float* dst) {
	CUDA_KERNEL_LOOP(index, n) {
		dst[index] = hipCrealf(a[index]);
	}
}

void caffe_gpu_real_C(const int N, const hipComplex* a, float* dst) {
	real_C_kernel<<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
			N, a, dst);
}


__global__ void add_C_kernel(const int n, const hipComplex* a,
		const hipComplex* b, hipComplex* dst) {
	CUDA_KERNEL_LOOP(index, n) {
		dst[index] = hipCaddf(a[index], b[index]);
	}
}

void caffe_add_mul_C(const int N, const hipComplex* a, const hipComplex* b,
		hipComplex* dst) {
	add_C_kernel<<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
			N, a, b, dst);
}
__global__ void sub_C_kernel(const int n, const hipComplex* a,
		const hipComplex* b, hipComplex* dst) {
	CUDA_KERNEL_LOOP(index, n) {
		dst[index] = hipCsubf(a[index], b[index]);
	}
}

void caffe_gpu_sub_C(const int N, const hipComplex* a, const hipComplex* b,
		hipComplex* dst) {
	sub_C_kernel<<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
			N, a, b, dst);
}
__global__ void mul_C_kernel(const int n, const hipComplex* a,
		const hipComplex* b, hipComplex* dst) {
	CUDA_KERNEL_LOOP(index, n) {
		dst[index] = hipCmulf(a[index], b[index]);
	}
}

void caffe_gpu_mul_C(const int N, const hipComplex* a, const hipComplex* b,
		hipComplex* dst) {
	mul_C_kernel<<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
			N, a, b, dst);
}

__global__ void mul_cjC_kernel(const int n, const hipComplex* a,
		const hipComplex* b, hipComplex* dst) {
	CUDA_KERNEL_LOOP(index, n) {
		dst[index] = hipCmulf(hipConjf(a[index]), b[index]);
	}
}

void caffe_gpu_mul_cjC(const int N, const hipComplex* a, const hipComplex* b,
		hipComplex* dst) {
	mul_cjC_kernel<<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
			N, a, b, dst);
}

__global__ void add_scalar_C_kernel(const int n, const hipComplex* a,
		const hipComplex alpha, hipComplex* dst) {
	CUDA_KERNEL_LOOP(index, n) {
		dst[index] = hipCaddf(a[index], alpha);
	}
}

void caffe_gpu_add_scalar_C(const int N, const hipComplex* a, const hipComplex alpha, 
		hipComplex* dst) {
	add_scalar_C_kernel<<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
			N, a, alpha, dst);
}

__global__ void div_C_kernel(const int n, const hipComplex* a,
		const hipComplex* b, hipComplex* y) {
	CUDA_KERNEL_LOOP(index, n) {
		y[index] = hipCdivf(a[index], b[index]);
	}
}

void caffe_gpu_div_C(const int N, const hipComplex* a,
		const hipComplex* b, hipComplex* dst) {
	// NOLINT_NEXT_LINE(whitespace/operators)
	div_C_kernel<<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
			N, a, b, dst);
}

}
