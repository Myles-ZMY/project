#include <algorithm>
#include <vector>

#include "caffe/layers/kcf_layer.hpp"
#include "caffe/util/math_functions.hpp"
// #include "hipfft/hipfft.h"

namespace caffe {

template <>
void KCFLayer<float>::Forward_gpu(
    const vector<Blob<float>*>& bottom, const vector<Blob<float>*>& top) {
  //hipMalloc((void**)& xf_, sizeof(hipfftComplex)*H_*(W_/2+1));
  hipfftReal* x = bottom[0]->mutable_gpu_data();
  hipfftComplex* xf = xf_.mutable_gpu_data();

  if (hipfftExecR2C(plan_, x, xf) != HIPFFT_SUCCESS){
    fprintf(stderr, "CUFFT Error: Unable to execute plan\n");
    return;
  }
  top[0]->mutable_cpu_data()[0] = float(1.0);
  return;
}

template <>
void KCFLayer<double>::Forward_gpu(
    const vector<Blob<double>*>& bottom, const vector<Blob<double>*>& top) {
  //hipMalloc((void**)& xf_d, sizeof(hipfftDoubleComplex)*H_*(W_/2+1));
  //hipfftDoubleReal* x = bottom[0]->mutable_gpu_data();
  //if (hipfftExecR2C(plan_, x, xf_d) != HIPFFT_SUCCESS){
  //  fprintf(stderr, "CUFFT Error: Unable to execute plan\n");
  //  return;
  //}
  top[0]->mutable_cpu_data()[0] = double(1.0);
  return;
}

template <typename Dtype>
void KCFLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  return;
}

INSTANTIATE_LAYER_GPU_FUNCS(KCFLayer);

}  // namespace caffe
