#include "hip/hip_runtime.h"
#include <math_functions.h>
#include <thrust/device_vector.h>
#include <thrust/functional.h>  // thrust::plus
#include <thrust/reduce.h>

#include "caffe/common.hpp"
#include "caffe/cukcf/cukcf.hpp"
#include "caffe/util/math_functions.hpp"
namespace caffe {

__global__ void mul_C_kernel(const int n, const hipComplex* a,
		const hipComplex* b, hipComplex* dst) {
	CUDA_KERNEL_LOOP(index, n) {
		dst[index] = hipCmulf(a[index], b[index]);
	}
}

void caffe_gpu_mul_C(const int N, const hipComplex* a, const hipComplex* b,
		hipComplex* dst) {
	mul_C_kernel<<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
			N, a, b, dst);
}

__global__ void mul_cjC_kernel(const int n, const hipComplex* a,
		const hipComplex* b, hipComplex* dst) {
	CUDA_KERNEL_LOOP(index, n) {
		dst[index] = hipCmulf(hipConjf(a[index]), b[index]);
	}
}

void caffe_gpu_mul_cjC(const int N, const hipComplex* a, const hipComplex* b,
		hipComplex* dst) {
	mul_cjC_kernel<<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
			N, a, b, dst);
}

__global__ void add_scalar_C_kernel(const int n, const hipComplex* a,
		const hipComplex alpha, hipComplex* dst) {
	CUDA_KERNEL_LOOP(index, n) {
		dst[index] = hipCaddf(a[index], alpha);
	}
}

void caffe_gpu_add_scalar_C(const int N, const hipComplex* a, const hipComplex alpha, 
		hipComplex* dst) {
	add_scalar_C_kernel<<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
			N, a, alpha, dst);
}

__global__ void div_C_kernel(const int n, const hipComplex* a,
		const hipComplex* b, hipComplex* y) {
	CUDA_KERNEL_LOOP(index, n) {
		y[index] = hipCdivf(a[index], b[index]);
	}
}

void caffe_gpu_div_C(const int N, const hipComplex* a,
		const hipComplex* b, hipComplex* dst) {
	// NOLINT_NEXT_LINE(whitespace/operators)
	div_C_kernel<<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
			N, a, b, dst);
}

}
