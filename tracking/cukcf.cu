#include <algorithm>
#include <iostream>
#include "cukcf.hpp"

#include "hip/hip_complex.h"
#include "hipfft/hipfft.h"

namespace CUKCF {


}

#define NX 8
#define BATCH 1

int main() {
  //std::cout << 0;
  hipComplex c;
  c = make_hipComplex(1.0, 0.0);

  hipfftHandle plan;
  hipfftComplex *data;
  hipMalloc((void**)&data, sizeof(hipfftComplex)*NX*BATCH);
//  std::cout << 1;
  data[0] = c;
//  std::cout << 2;
//  hipfftPlan1d(&plan, NX, HIPFFT_C2C, BATCH);
//  std::cout << 3;
//  hipfftExecC2C(plan, data, data, HIPFFT_FORWARD);
//  std::cout << 4;
  std::cout << "Hello, cuda." << c.x << std::endl;
  return 0;
}
