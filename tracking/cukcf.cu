#include <algorithm>
#include <iostream>
#include "cukcf.hpp"

#include "hip/hip_complex.h"
#include "hipfft/hipfft.h"

namespace CUKCF {


}

#define NX 8
#define BATCH 1

int main() {
  hipComplex c;
  c = make_hipComplex(1.0, 0.0);

  hipfftHandle plan;
  hipfftComplex *data;
  hipMalloc((void**)&data, sizeof(hipfftComplex)*NX*BATCH);
  hipfftPlan1d(&plan, NX, HIPFFT_C2C, BATCH);
  hipfftExecC2C(plan, data, data, HIPFFT_FORWARD);
  std::cout << "Hello, cuda." << c.x << std::endl;
  return 0;
}
